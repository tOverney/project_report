#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void loop_mult(unsigned int* a, unsigned int* b, int64_t* times_bef, int64_t* times_aft, int n) {
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int op_a = a[u];
    unsigned int op_b = b[u];

    times_bef[u] = clock64(); 
    for(int i=0; i<1000; i++) {
        asm volatile("mul.lo.u32 %0, %0, %1;" : "+r"(op_a) : "r"(op_b));
        /* 999 more multiplications */
    }
    timeis_aft[u] = clock64();

    a[u] = op_a;
}

int main(int argc, char** argv) {
    int n_threads = 1;
    int n_loops = 1;
    int n_grids = 1;

    if(argc == 2) {
        n_threads = atoi(argv[1]);
    }
    if(argc == 3) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
    }
    if(argc == 4) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
        n_grids = atoi(argv[3]);
    }

    dim3 dimBlock(n_threads, 1, 1);
    dim3 dimGrid(n_grids, 1, 1);

    unsigned int *a, *b;
    unsigned int *d_a, *d_b;
    int64_t *times_bef, *times_aft;
    int64_t *d_times_bef, *d_times_aft;

    a = (unsigned int *) malloc(n_threads*sizeof(unsigned int));
    b = (unsigned int *) malloc(n_threads*sizeof(unsigned int));
    times_bef = (int64_t *) malloc(n_threads*sizeof(int64_t));
    times_aft = (int64_t *) malloc(n_threads*sizeof(int64_t));

    for (int i = 0; i < n_threads; i++) {
        a[i] = (unsigned int) 3.1;
        b[i] = (unsigned int) 245321.0;
    }

    hipMalloc ( &d_a,n_threads*sizeof(unsigned int));
    hipMalloc ( &d_b,n_threads*sizeof(unsigned int));
    hipMalloc ( &d_times_bef,n_threads*sizeof(int64_t));
    hipMalloc ( &d_times_aft,n_threads*sizeof(int64_t));

    hipMemcpy(d_a, a, n_threads*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n_threads*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_times_bef, times_bef, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_times_aft, times_aft, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);

    loop_mult<<<dimGrid, dimBlock >>>(d_a, d_b, d_times_bef, d_times_aft, n_loops);
    
    hipMemcpy(a, d_a, n_threads*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(times_bef, d_times_bef, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(times_aft, d_times_aft, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);

    for( int i=0; i < n_threads; i++) {
        fprintf(stderr, "%lu %lu\n",times_bef[i],times_aft[i]); 
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_times_bef);
    hipFree(d_times_aft);
    free(a);
    free(b);
    free(times_aft);
    free(times_bef);
    return EXIT_SUCCESS;

}
