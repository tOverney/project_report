#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void loop_mult(DATA_TYPE* a, DATA_TYPE* b, int64_t* times_bef, int64_t* times_aft, int n) {
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    DATA_TYPE op_a = a[u];
    DATA_TYPE op_b = b[u];

    times_bef[u] = clock64();   
for(int i=0; i < n; i++) {
MEASUREMENT_TARGET}
    times_aft[u] = clock64();
    
    a[u] = op_a;
}

int main(int argc, char** argv) {
    int n_threads = 1;
    int n_loops = 1;
    int n_grids = 1;

    if(argc == 2) {
        n_threads = atoi(argv[1]);
    }
    if(argc == 3) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
    }
    if(argc == 4) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
        n_grids = atoi(argv[3]);
    }

    dim3 dimBlock(n_threads, 1, 1);
    dim3 dimGrid(n_grids, 1, 1);

    DATA_TYPE *a, *b;
    DATA_TYPE *d_a, *d_b;
    int64_t *times_bef, *times_aft;
    int64_t *d_times_bef, *d_times_aft;

    a = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));
    b = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));
    times_bef = (int64_t *) malloc(n_threads*sizeof(int64_t));
    times_aft = (int64_t *) malloc(n_threads*sizeof(int64_t));

    for (int i = 0; i < n_threads; i++) {
        a[i] = (DATA_TYPE) 3.1;
        b[i] = (DATA_TYPE) 2.0;
    }

    hipMalloc ( &d_a,n_threads*sizeof(DATA_TYPE));
    hipMalloc ( &d_b,n_threads*sizeof(DATA_TYPE));
    hipMalloc ( &d_times_bef,n_threads*sizeof(int64_t));
    hipMalloc ( &d_times_aft,n_threads*sizeof(int64_t));

    hipMemcpy(d_a, a, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_times_bef, times_bef, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_times_aft, times_aft, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);

    loop_mult<<<dimGrid, dimBlock >>>(d_a, d_b, d_times_bef, d_times_aft, n_loops);
    
    hipMemcpy(a, d_a, n_threads*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipMemcpy(times_bef, d_times_bef, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(times_aft, d_times_aft, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);

    for( int i=0; i < n_threads; i++) {
        printf("%lu %lu",times_bef[i],times_aft[i]); 
        if(i != n_threads-1) {
            printf(" \\n"); // allow us to avoid an empty cell in the python
        }
        // format is really important! 
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_times_bef);
    hipFree(d_times_aft);
    free(a);
    free(b);
    free(times_aft);
    free(times_bef);
    return EXIT_SUCCESS;

}
