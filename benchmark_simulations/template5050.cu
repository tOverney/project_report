#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void loop_mult(DATA_TYPE* a, DATA_TYPE* b, DATA_TYPE* c, DATA_TYPE* d, int64_t* times_bef, int64_t* times_aft, int n) {
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    DATA_TYPE op_a = a[u];
    DATA_TYPE op_b = b[u];
    DATA_TYPE op_c = c[u];
    DATA_TYPE op_d = d[u];

    times_bef[u] = clock();   
MEASUREMENT_TARGET
    time_aft = clock();
    times_aft[u] = (time_aft < times_bef[u]) ? time_aft + (0xffffffff - times_bef[u]) : time_aft;    

    a[u] = op_a;
	c[u] = op_c;
}

int main(int argc, char** argv) {
    int n_threads = 1;
    int n_loops = 1;
    int n_grids = 1;

    if(argc == 2) {
        n_threads = atoi(argv[1]);
    }
    if(argc == 3) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
    }
    if(argc == 4) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
        n_grids = atoi(argv[3]);
    }

    dim3 dimBlock(n_threads, 1, 1);
    dim3 dimGrid(n_grids, 1, 1);

    DATA_TYPE *a, *b, *c, *d;
    DATA_TYPE *d_a, *d_b, *d_c, *d_d;
    int64_t *times_bef, *times_aft;
    int64_t *d_times_bef, *d_times_aft;

    a = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));
    b = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));
	c = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));
    d = (DATA_TYPE *) malloc(n_threads*sizeof(DATA_TYPE));

    times_bef = (int64_t *) malloc(n_threads*sizeof(int64_t));
    times_aft = (int64_t *) malloc(n_threads*sizeof(int64_t));

    for (int i = 0; i < n_threads; i++) {
        a[i] = (DATA_TYPE) 3.1;
        b[i] = (DATA_TYPE) 2.0;
		c[i] = (DATA_TYPE) 3.1;
        d[i] = (DATA_TYPE) 2.0;
    }

    hipMalloc ( &d_a,n_threads*sizeof(DATA_TYPE));
    hipMalloc ( &d_b,n_threads*sizeof(DATA_TYPE));
	hipMalloc ( &d_c,n_threads*sizeof(DATA_TYPE));
    hipMalloc ( &d_d,n_threads*sizeof(DATA_TYPE));

    hipMalloc ( &d_times_bef,n_threads*sizeof(int64_t));
    hipMalloc ( &d_times_aft,n_threads*sizeof(int64_t));

    hipMemcpy(d_a, a, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, n_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    hipMemcpy(d_times_bef, times_bef, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_times_aft, times_aft, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);

    loop_mult<<<dimGrid, dimBlock >>>(d_a, d_b, d_c, d_d, d_times_bef, d_times_aft, n_loops);
    
    hipMemcpy(a, d_a, n_threads*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
	hipMemcpy(a, d_a, n_threads*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    hipMemcpy(times_bef, d_times_bef, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(times_aft, d_times_aft, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);

    for( int i=0; i < n_threads; i++) {
        fprintf(stderr, "%lu %lu",times_bef[i],times_aft[i]); 
        if(i != n_threads-1) {
            fprintf(stderr, " \n"); // allow us to avoid an empty cell in the python
        }
        // format is really important! 
    }
    
   hipFree(d_a);
    hipFree(d_b);
	hipFree(d_c);
    hipFree(d_d);

    hipFree(d_times_bef);
    hipFree(d_times_aft);
    free(a);
    free(b);
	free(c);
    free(d);
    free(times_aft);
    free(times_bef);
    return EXIT_SUCCESS;

}
