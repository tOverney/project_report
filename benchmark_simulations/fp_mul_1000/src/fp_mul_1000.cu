#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void loop_mult(float* a, float* b, int64_t* times_bef, int64_t* times_aft, int n) {
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t time_aft;
    float op_a = a[u];
    float op_b = b[u];

    times_bef[u] = clock();   
asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));

asm volatile("mul.f32 %0, %0, %1;" : "+f"(op_a) : "f"(op_b));


    time_aft = clock();
    times_aft[u] = (time_aft < times_bef[u]) ? time_aft + (0xffffffff - times_bef[u]) : time_aft;    

    a[u] = op_a;
}

int main(int argc, char** argv) {
    int n_threads = 1;
    int n_loops = 1;
    int n_grids = 1;

    if(argc == 2) {
        n_threads = atoi(argv[1]);
    }
    if(argc == 3) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
    }
    if(argc == 4) {
        n_threads = atoi(argv[1]);
        n_loops = atoi(argv[2]);
        n_grids = atoi(argv[3]);
    }

    dim3 dimBlock(n_threads, 1, 1);
    dim3 dimGrid(n_grids, 1, 1);

    float *a, *b;
    float *d_a, *d_b;
    int64_t *times_bef, *times_aft;
    int64_t *d_times_bef, *d_times_aft;

    a = (float *) malloc(n_threads*sizeof(float));
    b = (float *) malloc(n_threads*sizeof(float));
    times_bef = (int64_t *) malloc(n_threads*sizeof(int64_t));
    times_aft = (int64_t *) malloc(n_threads*sizeof(int64_t));

    for (int i = 0; i < n_threads; i++) {
        a[i] = (float) 3.1;
        b[i] = (float) 2.0;
    }

    hipMalloc ( &d_a,n_threads*sizeof(float));
    hipMalloc ( &d_b,n_threads*sizeof(float));
    hipMalloc ( &d_times_bef,n_threads*sizeof(int64_t));
    hipMalloc ( &d_times_aft,n_threads*sizeof(int64_t));

    hipMemcpy(d_a, a, n_threads*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n_threads*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_times_bef, times_bef, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_times_aft, times_aft, n_threads*sizeof(int64_t), hipMemcpyHostToDevice);

    loop_mult<<<dimGrid, dimBlock >>>(d_a, d_b, d_times_bef, d_times_aft, n_loops);
    
    hipMemcpy(a, d_a, n_threads*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(times_bef, d_times_bef, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemcpy(times_aft, d_times_aft, n_threads*sizeof(int64_t), hipMemcpyDeviceToHost);

    for( int i=0; i < n_threads; i++) {
        printf("%lu %lu",times_bef[i],times_aft[i]); 
        if(i != n_threads-1) {
            printf(" \n"); // allow us to avoid an empty cell in the python
        }
        // format is really important! 
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_times_bef);
    hipFree(d_times_aft);
    free(a);
    free(b);
    free(times_aft);
    free(times_bef);
    return EXIT_SUCCESS;

}
